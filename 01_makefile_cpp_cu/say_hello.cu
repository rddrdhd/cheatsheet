#include "hip/hip_runtime.h"
#include "say_hello.h"

__global__ 
void say_hello_kernel()
{
    int global_idx = blockIdx.x * blockDim.x + threadIdx.x;
    int n_threads_total = blockDim.x * gridDim.x;

    printf("Hello GPU world from thread %d/%d, block %d/%d, my global index is %d/%d\n", threadIdx.x, blockDim.x, blockIdx.x, gridDim.x, global_idx, n_threads_total);
}

void say_hello(){
    printf("Launching the kernel with 2 blocks, each with 4 threads\n");

    // launch the kernel
    say_hello_kernel<<< 2, 4 >>>();

    printf("Kernel was launched, waiting for its completion\n");
   
    // wait for the kernel to finish
    hipDeviceSynchronize();

    printf("Kernel execution completed\n");
}